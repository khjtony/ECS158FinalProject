#include <vector>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <math.h>
#include <cstring>
#include <hip/hip_runtime.h>

#define ITER_LIMIT 500
#define MATRIX_MAX 10

using namespace std;


void nmfgpu(float *a, int r, int c, int k, int niters, float *w, float *h);
void matrix_print(float *a, int r, int c);
void nmf_seed(float *out, float *a, int r, int c, int k);
void surface_matrix_multi(float *out, float *a, float *b, int r1, int c1, int r2, int c2);


__global__ void kernel(float *a, int r, int c, int k, int niters, float *w,
 float *h, float* wt, float* ht, float* wta, float* wtw, float* wtwh, float* hdiv, float* aht,
    	float* wh, float* whht, float* wdiv, int total);


__device__ void matrix_multi(float *out, float *a, float *b, int r1, int c1, int r2, int c2, int total);
__device__ void matrix_trans(float *out, float *a, int r, int c, int total);
__device__ float matrix_findmax(float *a, int r, int c, int total);
__device__ float matrix_distance(float *a, float *b, int r, int c, int total);
__device__ void matrix_elemproduct(float* out, float *a, float *b, int r, int c, int total);
__device__ void matrix_elemdivison(float* out, float *a, float *b, int r ,int c, int total);


void surface_matrix_multi(float *out, float *a, float *b, int r1, int c1, int r2, int c2){
    if (c1!=r2){
    	return;
    }
    // int total = omp_get_thread_num();
    // int total = omp_get_num_threads();
    int nth = 0;
    int total = 1;
    int division = r1/total;
    for (int row = (division*total); row < (nth==(total-1) ? r1 : (division*nth+division)); row++) {
        for (int col = 0; col < c2; col++) {
            out[row*c2+col] = 0;
            for (int inner = 0; inner < c1; inner++) {
                out[row*c2+col] += a[row*c1+inner] * b[inner*c2+col]*1.0;
                // printf("%d %d :%.2f += %.2f\n", 
                //     row, col, out[row*c2+col]-a[row*c1+inner] * b[inner*c2+col]*1.0, a[row*c1+inner] * b[inner*c2+col]*1.0);
            }
        }
    }
}




void nmfgpu(float *a, int r, int c, int k, int niters, float *w, float *h){
	// setup cuda
    float *dm;
    float *dm_w;
    float *dm_h;
    int msize = r*c*sizeof(float);
    int msize_w = r*k*sizeof(float);
    int msize_h = k*c*sizeof(float);
    float *dm_wt = new float[r*k];
    float *dm_ht = new float[c*k];
    float *dm_wta = new float[k*c];
    float *dm_wtw = new float[k*k];
    float *dm_wtwh = new float[k*c];
    float *dm_hdiv = new float[k*c];
    float *dm_aht = new float[r*k];
    float *dm_wh = new float[r*c];
    float *dm_whht = new float[r*k];
    float *dm_wdiv = new float[r*k];
    nmf_seed(w, a, r, c, k);
    memcpy(h, w, sizeof(float)*r*k);
    hipMalloc((void**)&dm, msize);
    hipMalloc((void**)&dm_h, msize_h);
    hipMalloc((void**)&dm_w, msize_w);
    hipMalloc((void**)&dm_wt, r*k*sizeof(float));
    hipMalloc((void**)&dm_ht, c*k*sizeof(float));
    hipMalloc((void**)&dm_wta, k*c*sizeof(float));
    hipMalloc((void**)&dm_wtw, k*k*sizeof(float));
    hipMalloc((void**)&dm_wtwh, k*c*sizeof(float));
    hipMalloc((void**)&dm_hdiv, k*c*sizeof(float));
    hipMalloc((void**)&dm_aht, r*k*sizeof(float));
    hipMalloc((void**)&dm_wh, r*c*sizeof(float));
    hipMalloc((void**)&dm_whht,r*k*sizeof(float));
    hipMalloc((void**)&dm_wdiv,r*k*sizeof(float));

    int total = 4;
    dim3 dimGrid(1,1);
    dim3 dimBlock(total, 1, 1);

    hipMemcpy(dm, a, msize, hipMemcpyHostToDevice);
    kernel<<<dimGrid, dimBlock>>>(dm, r, c, k, niters , dm_w, dm_h,
    	dm_wt, dm_ht, dm_wta, dm_wtw, dm_wtwh, dm_hdiv, dm_aht,
    	dm_wh, dm_whht, dm_wdiv, total);

    hipMemcpy(w, dm_w, msize_w, hipMemcpyDeviceToHost);
    hipMemcpy(h, dm_h, msize_h, hipMemcpyDeviceToHost);
    hipFree(dm);
	hipFree(dm_h);
	hipFree(dm_w);
	hipFree(dm_wt);
	hipFree(dm_ht);
	hipFree(dm_wta);
	hipFree(dm_wtw);
	hipFree(dm_wtwh);
	hipFree(dm_hdiv);
	hipFree(dm_aht);
	hipFree(dm_wh);
	hipFree(dm_whht);
	hipFree(dm_wdiv);
}




__global__ void kernel(float *a, int r, int c, int k, int niters, float *w, float *h,
	float* wt, float* ht, float* wta, float* wtw, float* wtwh, float* hdiv, float* aht,
    	float* wh, float* whht, float* wdiv, int total){



    // calculate h
    // nominator
    // printf("NMF: 1\n");
    for (int i=0;i<niters;i++){
	    matrix_trans(wt, w, r, k, total);
	    matrix_multi(wta, wt, a, k, r, r, c, total);
	    // denominator
	    // printf("NMF: 2\n");
	    matrix_multi(wtw, wt, w, k, r, r, k, total);
	    matrix_multi(wtwh, wtw, h, k, k, k, c, total);
	    // new h
	    // printf("NMF: 3\n");
	    matrix_elemdivison(hdiv, wta, wtwh, k, c, total);
	    matrix_elemproduct(h, h, hdiv, k, c, total);
	    // calcualte w
	    // nominator
	    // printf("NMF: 4\n");
	    matrix_trans(ht, h, k, c, total);
	    matrix_multi(aht, a, ht, r, c, c, k, total);
	    // calculate w
	    // printf("NMF: 5\n");
	    matrix_multi(wh, w, h, r, k, k, c, total);
	    matrix_multi(whht, wh, ht, r, c, c, k, total);
	    matrix_elemdivison(wdiv, aht, whht, r, k, total);
	    // new w
	    // printf("NMF: 6\n");
	    matrix_elemproduct(w, w, wdiv, r, k, total);
    }
}



__device__ void matrix_multi(float *out, float *a, float *b, int r1, int c1, int r2, int c2, int total){
    if (c1!=r2){
    	return;
    }
    // int total = omp_get_thread_num();
    // int total = omp_get_num_threads();
    int nth = threadIdx.x;
    int division = r1/total;
    for (int row = (division*nth); row < (nth==(total-1) ? r1 : (division*nth+division)); row++) {
        for (int col = 0; col < c2; col++) {
            out[row*c2+col] = 0;
            for (int inner = 0; inner < c1; inner++) {
                out[row*c2+col] += a[row*c1+inner] * b[inner*c2+col]*1.0;
                // printf("%d %d :%.2f += %.2f\n", 
                //     row, col, out[row*c2+col]-a[row*c1+inner] * b[inner*c2+col]*1.0, a[row*c1+inner] * b[inner*c2+col]*1.0);
            }
        }
    }
}

__device__ void matrix_trans(float *out, float *a, int r, int c, int total){
    // This trans function can be boosted up by multi thread
    
 
    // int total = omp_get_thread_num();
    // int total = omp_get_num_threads();
    int nth = threadIdx.x;
    int division = c/total;
    for (int j=(division*nth);j<(nth==(total-1) ? c : (division*nth+division));j++){
        for (int i=0;i<r;i++){
            out[j*r+i]=a[i*c+j];
        }
    }
}

void matrix_print(float *a, int r, int c){
    for(int i=0;i<r;i++){
        for (int j=0;j<c;j++){
            printf("%.2f\t", a[i*c+j]);

        }
        printf("\n");
    }
}

__device__ float matrix_findmax(float *a, int r, int c, int total){
    // this function can be boosted by find max in row/col and compare them
    if (r==0 && c==0){
        return 0;
    }
    float max = a[0];
    for (int i = 0;i<r*c;i++){
        if (a[i]>max){
            max=a[i];
        }
    }
    return max;
}

void nmf_seed(float *out, float *a, int r, int c, int k){
    // setup generator
   	srand(time(NULL));
    for (int i=0;i<r*k;i++){
        out[i] = (float)(MATRIX_MAX*((rand()%100/100.0)));
    }

}


__device__ float matrix_distance(float *a, float *b, int r, int c, int total){
    float distance=0;
    for (int i=0;i<r*c;i++){
        distance += abs(a[i]-b[i])*abs(a[i]-b[i]);
    }
    
    return distance;
}


__device__ void matrix_elemproduct(float* out, float *a, float *b, int r, int c, int total){
    // int total = omp_get_thread_num();
    // int total = omp_get_num_threads();
    int nth = threadIdx.x;
    int division = r*c/total;
    // printf("Its %d/%d\n", total, total);
    for (int i=(division*nth);i<(nth==(total-1) ? r*c : (division*nth+division));i++){
            out[i]=1.0*a[i]*b[i];
    }
}


__device__ void matrix_elemdivison(float* out, float *a, float *b, int r ,int c, int total){
    // int total = omp_get_thread_num();
    // int total = omp_get_num_threads();
    int nth = threadIdx.x;
    int division = r*c/total;
    for (int i=(division*nth);i<(nth==(total-1) ? r*c : (division*nth+division));i++){
            if (b[i]==0){
            	return;
            }
            out[i]=1.0*a[i]/b[i];
            // printf("Now is %d\n", i);
    }
}


int main(int argc, char const *argv[])
{
    // omp_set_num_threads(8);
    /* code */
    // float a[12]={2, 3, 4, 1, 2, 10, 11, 32,
    //             3.1, 4.1, 31, 0.2};
    // float b[12]={1, 2, 1, 3, 1, 4, 1, 5,
    //             1, 6, 1, 7};


    // matrix for nmf. rxc (row, col)
    // int r = 300;
    // int c = 200;
    // int k = 150; 
    int r = 5;
    int c = 4;
    int k = 3;
    float B[20]={1, 2, 3, 4,
            5, 6, 7, 8,
            9, 10, 11, 12,
            13, 14, 15, 16,
            17, 18, 19, 20};


    

    // matrix generator
	srand(time(NULL));
    float A[r*c];
    for (int i=0;i<r;i++){
        for (int j=0;j<c;j++){
            A[i*c+j]=(float)(1.0*MATRIX_MAX*((float)(rand()%100/100.0)));
        }
    }
 



    // nmf using originall 
    float w[r*k];
    float h[k*c];


    // double time_begin=get_wall_time();
    for (int i=0;i<1;i++){
        nmfgpu(A, r, c, k, 500, w, h);
        printf("It's run: \t%d\n", i);
    }
    // double time_end=get_wall_time();
    // printf("Matrix w is: \n");
    // matrix_print(w, 5, 3);
    // printf("Matrix h is: \n");
    // matrix_print(h, 3, 4);

    printf("Matrix A is: \n");
    matrix_print(A, r, c);
    surface_matrix_multi(A, w, h, r, k, k, c);
    printf("Matrix w*h is: \n");
    matrix_print(A, r, c);


    return 0;
}