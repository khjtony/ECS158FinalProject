#include "hip/hip_runtime.h"

__global__ void kernel(float *A, float *B, int row, int col, float *out){
    int nth = threadIdx.x;
    int division = r1/total;
    int i=0;
    int j=0;
    int inner=0;
    for (i = (division*nth); i < (nth==(total-1) ? row : (division*nth+division)); i++) {
        for (j = 0; j < col; j++) {
            out[i*col+j] = 0;
            for (inner = 0; inner < col; inner++) {
                out[i*col+j] += a[i*col+inner] * b[inner*col+j]*1.0;
            }
        }
    }
}

void mmul_cuda(float *A, float *B, int row, int col, float *out){
    float *dA, *dB, *dout;
    int msize = row*col*sizeof(float);
    hipMalloc((void**)&dA, msize);
    hipMalloc((void**)&dB, msize);
    hipMalloc((void**)&dm, msize);
    hipMemcpy(dA, A, msize, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, msize, hipMemcpyHostToDevice);

    int blockSize = 4;  // use 4 blocks
    dim3 dimGrid(blockSize, 1);
    dim3 dimBlock(1,1,1);

    kernel<<<dimGrid, dimBlock>>>(dA, dB, row, col, dm);

    hipMemcpy(out, dm, msize, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dm);

    return;

}