hipMalloc(A_device)  // Allocate Memory on Device
hipMemcpy(A_host, A_device, hipMemcpyHostToDevice)  // Initialize A with the Data from Host
hipMemcpy(A_host, A_device, hipMemcpyDeviceToHost)  // Copy the Resultant Data back to Host
hipFree(A_device)  // Free Device Memory
